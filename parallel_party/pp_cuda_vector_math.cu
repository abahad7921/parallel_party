#include "hip/hip_runtime.h"







//pp_cuda_vector_math
//--------------------------------------------------
/*
Implementation of cuda vector math and kernels. Encapsulated to be compiled with nvcc
*/


//Preprocessor
//--------------------------------------------------




//Include
//--------------------------------------------------
//cuda
#include "hip/hip_vector_types.h"

//own
#include "pp_cuda_vector_math.h"






//kernels
//--------------------------------------------------
namespace kernel
{

	//sine_deform
	__global__ void sine_deform(float* x, float* y, float* z, float* result_x, float* result_y, float* result_z, float random, int size)
	{
		//thread_index
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		//check and add
		if(index < size)
		{
			result_x[index] = x[index];
			result_y[index] = sin(x[index] + z[index] + random);
			result_z[index] = z[index];
		};
		
	};

	//print_pointer
	__global__ void print_pointer(float* ptr, int size)
	{
		for(int index = 0; index < size; index++)
		{
			printf("Value: %d \n", ptr[index]);
		};
	};

};



//cuda
//--------------------------------------------------

//is_cuda_available
int cuda::is_cuda_available()
{
	int deviceCount = 0;
	hipError_t cudaErrorId;
	cudaErrorId = hipGetDeviceCount(&deviceCount);
	return deviceCount;
};


//get_device_attributes
std::string cuda::get_device_attributes()
{
	
	//deviceCount
	int deviceCount = is_cuda_available();

	//no device available
	if(!deviceCount)
		return "No device available";
	
	//cudaDeviceProperties
	hipDeviceProp_t cudaDeviceProperties;

	//get device properties
	hipGetDeviceProperties(&cudaDeviceProperties, deviceCount - 1);

	//cudaDeviceProperties_str
	std::string cudaDeviceProperties_str = get_dev_properties_str(&cudaDeviceProperties);

	return cudaDeviceProperties_str;
};


//get_dev_properties_str
std::string cuda::get_dev_properties_str(void* ptr)
{
	
	//hipDeviceProp_t*
	hipDeviceProp_t* devProp = static_cast<hipDeviceProp_t*>(ptr);

	//format_string
	boost::format format_string("\n\nCUDA device information:\n"
								"--------------------------------------\n"
								"Major revision number:         %d\n"
								"Minor revision number:         %d\n"
								"Name:                          %s\n"
								
								"Total global memory:           %u\n"
								"Total shared memory per block: %u\n"
								"Total registers per block:     %d\n"
								"Warp size:                     %d\n"
								"Maximum memory pitch:          %u\n"
								
								"Maximum threads per block:     %d\n"
								"Maximum X dimension of block:  %d\n"
								"Maximum Y dimension of block:  %d\n"
								"Maximum Z dimension of block:  %d\n"

								"Maximum X dimension of grid:   %d\n"
								"Maximum Y dimension of grid:   %d\n"
								"Maximum Z dimension of grid:   %d\n"

								"Clock rate:                    %d\n"
								"Total constant memory:         %u\n"
								"Texture alignment:             %u\n"
								"Concurrent copy and execution: %s\n"
								"Number of multiprocessors:     %d\n"
								"Kernel execution timeout:      %s\n"
								"--------------------------------------\n\n"
								);

	//format
	format_string % devProp->major 
		% devProp->minor 
		% devProp->name
		
		% devProp->totalGlobalMem
		% devProp->sharedMemPerBlock
		% devProp->regsPerBlock
		% devProp->warpSize
		% devProp->memPitch

		% devProp->maxThreadsPerBlock
		% devProp->maxThreadsDim[0]
		% devProp->maxThreadsDim[1]
		% devProp->maxThreadsDim[2]

		% devProp->maxGridSize[0]
		% devProp->maxGridSize[1]
		% devProp->maxGridSize[2]

		% devProp->clockRate
		% devProp->totalConstMem
		% devProp->textureAlignment
		% (devProp->deviceOverlap ? "Yes" : "No")
		% devProp->multiProcessorCount
		% (devProp->kernelExecTimeoutEnabled ? "Yes" : "No")
		;

	return format_string.str();
}







//Pp_gpu_vector_math_optimized
//--------------------------------------------------

//sine_deform
void cuda::sine_deform(float*& ptr_float_x, float*& ptr_float_y, float*& ptr_float_z,
						float*& ptr_result_float_x, float*& ptr_result_float_y, float*& ptr_result_float_z,
						float random, int size)
{
	std::cout << "Execute sine_deform kernel" << std::endl;
	

	//Create and allocate device ptr
	//--------------------------------------------------
	//size_t
	size_t array_size = size * sizeof(float);

	//create device pointer
	float* ptr_d_float_x;
	float* ptr_d_float_y;
	float* ptr_d_float_z;

	float* ptr_d_result_float_x;
	float* ptr_d_result_float_y;
	float* ptr_d_result_float_z;
	
	//allocate device memory
	hipMalloc(&ptr_d_float_x, array_size);
	hipMalloc(&ptr_d_float_y, array_size);
	hipMalloc(&ptr_d_float_z, array_size);

	hipMalloc(&ptr_d_result_float_x, array_size);
	hipMalloc(&ptr_d_result_float_y, array_size);
	hipMalloc(&ptr_d_result_float_z, array_size);

	
	
	//memcpy to device
	//--------------------------------------------------
	
	//memcpy
	hipMemcpy(ptr_d_float_x, ptr_float_x, array_size, hipMemcpyHostToDevice);
	hipMemcpy(ptr_d_float_y, ptr_float_y, array_size, hipMemcpyHostToDevice);
	hipMemcpy(ptr_d_float_z, ptr_float_z, array_size, hipMemcpyHostToDevice);

	
	
	
	//kernel
	//--------------------------------------------------

	//thread_count
	int thread_count = 512;
	//grid_dim
	dim3 grid_dim((size + (thread_count - 1)) / thread_count);
	std::cout << ((size + (thread_count - 1)) / thread_count) << std::endl;
	
	//kernel::sine_deform
	kernel::sine_deform<<<grid_dim, thread_count>>> (ptr_d_float_x, ptr_d_float_y, ptr_d_float_z,
									ptr_d_result_float_x, ptr_d_result_float_y, ptr_d_result_float_z,
									random,
									size
									);




	//copy to host
	//--------------------------------------------------

	//memcpy
	hipMemcpy(ptr_result_float_x, ptr_d_result_float_x, array_size, hipMemcpyDeviceToHost);
	hipMemcpy(ptr_result_float_y, ptr_d_result_float_y, array_size, hipMemcpyDeviceToHost);
	hipMemcpy(ptr_result_float_z, ptr_d_result_float_z, array_size, hipMemcpyDeviceToHost);

	
	//free memory
	//--------------------------------------------------

	hipFree(ptr_d_float_x);
	hipFree(ptr_d_float_y);
	hipFree(ptr_d_float_z);
	
	hipFree(ptr_d_result_float_x);
	hipFree(ptr_d_result_float_x);
	hipFree(ptr_d_result_float_x);


};